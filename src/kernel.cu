
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#undef  assert
#define assert(arg)
#endif

__constant__ const float* sfe;
__constant__ const float* sfd;
__constant__ int sfs;
__constant__ float3 cr0;
__constant__ float3 cr1;
__constant__ int3 npr;
__constant__ float gri;
__constant__ const float* mps[15];
__constant__ int nbi;
__constant__ unsigned long sed;
__constant__ float* __restrict__ s0e;
__constant__ const int* __restrict__ lig;

extern __shared__ int shared[];

__device__  __noinline__// __forceinline__
bool evaluate(float* e, float* g, float* a, float* q, float* c, float* d, float* f, float* t, const float* x, const int nf, const int na, const int np, const float eub)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int gds = blockDim.x * gridDim.x;
	const int gd3 = 3 * gds;
	const int gd4 = 4 * gds;

	const int* act = shared;
	const int* beg = act + nf;
	const int* end = beg + nf;
	const int* nbr = end + nf;
	const int* prn = nbr + nf;
	const float* yy0 = (float*)(prn + nf);
	const float* yy1 = yy0 + nf;
	const float* yy2 = yy1 + nf;
	const float* xy0 = yy2 + nf;
	const float* xy1 = xy0 + nf;
	const float* xy2 = xy1 + nf;
	const int* brs = (int*)(xy2 + nf);
	const float* co0 = (float*)(brs + nf - 1);
	const float* co1 = co0 + na;
	const float* co2 = co1 + na;
	const int* xst = (int*)(co2 + na);
	const int* ip0 = xst + na;
	const int* ip1 = ip0 + np;
	const int* ipp = ip1 + np;

	float y, y0, y1, y2, v0, v1, v2, c0, c1, c2, e000, e100, e010, e001, a0, a1, a2, ang, sng, r0, r1, r2, r3, vs, dr, f0, f1, f2, t0, t1, t2, d0, d1, d2;
	float q0, q1, q2, q3, q00, q01, q02, q03, q11, q12, q13, q22, q23, q33, m0, m1, m2, m3, m4, m5, m6, m7, m8;
	int i, j, k, b, w, i0, i1, i2, k0, k1, k2, z;
	const float* map;

	// Apply position, orientation and torsions.
	c[i  = gid] = x[k  = gid];
	c[i += gds] = x[k += gds];
	c[i += gds] = x[k += gds];
	q[i  = gid] = x[k += gds];
	q[i += gds] = x[k += gds];
	q[i += gds] = x[k += gds];
	q[i += gds] = x[k += gds];
	y = 0.0f;
	for (k = 0, b = 0, w = 6 * gds + gid; k < nf; ++k)
	{
		// Load rotorY from memory into registers.
		y0 = c[i0  = beg[k] * gd3 + gid];
		y1 = c[i0 += gds];
		y2 = c[i0 += gds];

		// Translate orientation of active frames from quaternion into 3x3 matrix.
		if (act[k])
		{
			q0 = q[k0  = k * gd4 + gid];
			q1 = q[k0 += gds];
			q2 = q[k0 += gds];
			q3 = q[k0 += gds];
			//assert(fabs(q0*q0 + q1*q1 + q2*q2 + q3*q3 - 1.0f) < 2e-3f);
			q00 = q0 * q0;
			q01 = q0 * q1;
			q02 = q0 * q2;
			q03 = q0 * q3;
			q11 = q1 * q1;
			q12 = q1 * q2;
			q13 = q1 * q3;
			q22 = q2 * q2;
			q23 = q2 * q3;
			q33 = q3 * q3;
			m0 = q00 + q11 - q22 - q33;
			m1 = 2 * (q12 - q03);
			m2 = 2 * (q02 + q13);
			m3 = 2 * (q03 + q12);
			m4 = q00 - q11 + q22 - q33;
			m5 = 2 * (q23 - q01);
			m6 = 2 * (q13 - q02);
			m7 = 2 * (q01 + q23);
			m8 = q00 - q11 - q22 + q33;
		}

		// Evaluate c and d of frame atoms. Aggregate e into y.
		for (i = beg[k], z = end[k]; i < z; ++i)
		{
			i0 = i * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;

			// The first atom of a frame is assumed to be its rotor Y.
			if (i == beg[k])
			{
				c0 = y0;
				c1 = y1;
				c2 = y2;
			}
			else
			{
				// Calculate coordinate from transformation matrix and offset.
				v0 = co0[i];
				v1 = co1[i];
				v2 = co2[i];
				c0 = y0 + m0 * v0 + m1 * v1 + m2 * v2;
				c1 = y1 + m3 * v0 + m4 * v1 + m5 * v2;
				c2 = y2 + m6 * v0 + m7 * v1 + m8 * v2;

				// Store coordinate from registers into memory.
				c[i0] = c0;
				c[i1] = c1;
				c[i2] = c2;
			}

			// Penalize out-of-box case.
			if (c0 < cr0.x || cr1.x <= c0 || c1 < cr0.y || cr1.y <= c1 || c2 < cr0.z || cr1.z <= c2)
			{
				y += 10.0f;
				d[i0] = 0.0f;
				d[i1] = 0.0f;
				d[i2] = 0.0f;
				continue;
			}

			// Find the index of the current coordinate
			k0 = static_cast<int>((c0 - cr0.x) * gri);
			k1 = static_cast<int>((c1 - cr0.y) * gri);
			k2 = static_cast<int>((c2 - cr0.z) * gri);
			//assert(k0 + 1 < npr.x);
			//assert(k1 + 1 < npr.y);
			//assert(k2 + 1 < npr.z);
			k0 = npr.x * (npr.y * k2 + k1) + k0;

			// Retrieve the grid map and lookup the value
			 map = mps[xst[i]];
			e000 = map[k0];
			e100 = map[k0 + 1];
			e010 = map[k0 + npr.x];
			e001 = map[k0 + npr.x * npr.y];
			y += e000;
			d[i0] = (e100 - e000) * gri;
			d[i1] = (e010 - e000) * gri;
			d[i2] = (e001 - e000) * gri;
		}
		for (j = 0, z = nbr[k]; j < z; ++j)
		{
			i = brs[b++];
			i0 = beg[i] * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;
			c[i0] = y0 + m0 * yy0[i] + m1 * yy1[i] + m2 * yy2[i];
			c[i1] = y1 + m3 * yy0[i] + m4 * yy1[i] + m5 * yy2[i];
			c[i2] = y2 + m6 * yy0[i] + m7 * yy1[i] + m8 * yy2[i];

			// Skip inactive BRANCH frame
			if (!act[i]) continue;

			// Update a of BRANCH frame
			a0 = m0 * xy0[i] + m1 * xy1[i] + m2 * xy2[i];
			a1 = m3 * xy0[i] + m4 * xy1[i] + m5 * xy2[i];
			a2 = m6 * xy0[i] + m7 * xy1[i] + m8 * xy2[i];
			//assert(fabs(a0*a0 + a1*a1 + a2*a2 - 1.0f) < 2e-3f);
			a[k0  = i * gd3 + gid] = a0;
			a[k0 += gds] = a1;
			a[k0 += gds] = a2;

			// Update q of BRANCH frame
			ang = x[w += gds] * 0.5f;
//			sng = sinf(ang);
//			r0 = cosf(ang);
			sincosf(ang, &sng, &r0);
//			sincospif(ang, &sng, &r0);
			r1 = sng * a0;
			r2 = sng * a1;
			r3 = sng * a2;
			q00 = r0 * q0 - r1 * q1 - r2 * q2 - r3 * q3;
			q01 = r0 * q1 + r1 * q0 + r2 * q3 - r3 * q2;
			q02 = r0 * q2 - r1 * q3 + r2 * q0 + r3 * q1;
			q03 = r0 * q3 + r1 * q2 - r2 * q1 + r3 * q0;
			//assert(fabs(q00*q00 + q01*q01 + q02*q02 + q03*q03 - 1.0f) < 2e-3f);
			q[k0  = i * gd4 + gid] = q00;
			q[k0 += gds] = q01;
			q[k0 += gds] = q02;
			q[k0 += gds] = q03;
		}
	}
	//assert(b == nf - 1);
//	assert(w == nv * gds + gid);
	//assert(k == nf);

	// Calculate intra-ligand free energy.
	for (i = 0; i < np; ++i)
	{
		i0 = ip0[i] * gd3 + gid;
		i1 = i0 + gds;
		i2 = i1 + gds;
		k0 = ip1[i] * gd3 + gid;
		k1 = k0 + gds;
		k2 = k1 + gds;
		v0 = c[k0] - c[i0];
		v1 = c[k1] - c[i1];
		v2 = c[k2] - c[i2];
		vs = v0*v0 + v1*v1 + v2*v2;
		if (vs < 64.0f)
		{
			j = ipp[i] + static_cast<int>(sfs * vs);
			y += sfe[j];
			dr = sfd[j];
			d0 = dr * v0;
			d1 = dr * v1;
			d2 = dr * v2;
			d[i0] -= d0;
			d[i1] -= d1;
			d[i2] -= d2;
			d[k0] += d0;
			d[k1] += d1;
			d[k2] += d2;
		}
	}

	// If the free energy is no better than the upper bound, refuse this conformation.
	if (y >= eub) return false;

	// Store e from register into memory.
	e[gid] = y;

	// Calculate and aggregate the force and torque of BRANCH frames to their parent frame.
	f[k0 = gid] = 0.0f;
	t[k0] = 0.0f;
	for (i = 1, z = 3 * nf; i < z; ++i)
	{
		f[k0 += gds] = 0.0f;
		t[k0] = 0.0f;
	}
//	assert(w == nv * gds + gid);
	//assert(k == nf);
	while (k)
	{
		--k;

		// Load f, t and rotorY from memory into register
		k0 = k * gd3 + gid;
		k1 = k0 + gds;
		k2 = k1 + gds;
		f0 = f[k0];
		f1 = f[k1];
		f2 = f[k2];
		t0 = t[k0];
		t1 = t[k1];
		t2 = t[k2];
		y0 = c[i0  = beg[k] * gd3 + gid];
		y1 = c[i0 += gds];
		y2 = c[i0 += gds];

		// Aggregate frame atoms.
		for (i = beg[k], z = end[k]; i < z; ++i)
		{
			i0 = i * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;
			d0 = d[i0];
			d1 = d[i1];
			d2 = d[i2];

			// The derivatives with respect to the position, orientation, and torsions
			// would be the negative total force acting on the ligand,
			// the negative total torque, and the negative torque projections, respectively,
			// where the projections refer to the torque applied to the branch moved by the torsion,
			// projected on its rotation axi
			f0 += d0;
			f1 += d1;
			f2 += d2;
			if (i == beg[k]) continue;

			v0 = c[i0] - y0;
			v1 = c[i1] - y1;
			v2 = c[i2] - y2;
			t0 += v1 * d2 - v2 * d1;
			t1 += v2 * d0 - v0 * d2;
			t2 += v0 * d1 - v1 * d0;
		}

		if (k)
		{
			// Save the aggregated torque of active BRANCH frames to g.
			if (act[k])
			{
				g[w -= gds] = t0 * a[k0] + t1 * a[k1] + t2 * a[k2]; // dot product
			}

			// Aggregate the force and torque of current frame to its parent frame.
			k0 = prn[k] * gd3 + gid;
			k1 = k0 + gds;
			k2 = k1 + gds;
			f[k0] += f0;
			f[k1] += f1;
			f[k2] += f2;
			v0 = y0 - c[i0  = beg[prn[k]] * gd3 + gid];
			v1 = y1 - c[i0 += gds];
			v2 = y2 - c[i0 += gds];
			t[k0] += t0 + v1 * f2 - v2 * f1;
			t[k1] += t1 + v2 * f0 - v0 * f2;
			t[k2] += t2 + v0 * f1 - v1 * f0;
		}
	}
	//assert(w == 6 * gds + gid);

	// Save the aggregated force and torque of ROOT frame to g.
	g[i0  = gid] = f0;
	g[i0 += gds] = f1;
	g[i0 += gds] = f2;
	g[i0 += gds] = t0;
	g[i0 += gds] = t1;
	g[i0 += gds] = t2;
	return true;
}

extern "C" __global__
//__launch_bounds__(maxThreadsPerBlock, minBlocksPerMultiprocessor)
void monte_carlo(const int nv, const int nf, const int na, const int np)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int gds = blockDim.x * gridDim.x;
	const int nls = 5; // Number of line search trials for determining step size in BFGS
	const float eub = 40.0f * na; // A conformation will be droped if its free energy is not better than e_upper_bound.
	float* s0x = s0e + gds;
	float* s0g = s0x + (nv + 1) * gds;
	float* s0a = s0g + nv * gds;
	float* s0q = s0a + 3 * nf * gds;
	float* s0c = s0q + 4 * nf * gds;
	float* s0d = s0c + 3 * na * gds;
	float* s0f = s0d + 3 * na * gds;
	float* s0t = s0f + 3 * nf * gds;
	float* s1e = s0t + 3 * nf * gds;
	float* s1x = s1e + gds;
	float* s1g = s1x + (nv + 1) * gds;
	float* s1a = s1g + nv * gds;
	float* s1q = s1a + 3 * nf * gds;
	float* s1c = s1q + 4 * nf * gds;
	float* s1d = s1c + 3 * na * gds;
	float* s1f = s1d + 3 * na * gds;
	float* s1t = s1f + 3 * nf * gds;
	float* s2e = s1t + 3 * nf * gds;
	float* s2x = s2e + gds;
	float* s2g = s2x + (nv + 1) * gds;
	float* s2a = s2g + nv * gds;
	float* s2q = s2a + 3 * nf * gds;
	float* s2c = s2q + 4 * nf * gds;
	float* s2d = s2c + 3 * na * gds;
	float* s2f = s2d + 3 * na * gds;
	float* s2t = s2f + 3 * nf * gds;
	float* bfh = s2t + 3 * nf * gds;
	float* bfp = bfh + (nv*(nv+1)>>1) * gds;
	float* bfy = bfp + nv * gds;
	float* bfm = bfy + nv * gds;
	float rd0, rd1, rd2, rd3, rst;
	float sum, pg1, pga, pgc, alp, pg2, pr0, pr1, pr2, nrm, ang, sng, pq0, pq1, pq2, pq3, s1xq0, s1xq1, s1xq2, s1xq3, s2xq0, s2xq1, s2xq2, s2xq3, bpi;
	float yhy, yps, ryp, pco, bpj, bmj, ppj;
	int g, i, j, o0, o1, o2;
	hiprandState crs;

	// Load ligand into external shared memory.
	g = 11 * nf + nf - 1 + 4 * na + 3 * np;
	o0 = threadIdx.x;
	for (i = 0, j = (g - 1) / blockDim.x; i < j; ++i)
	{
		shared[o0] = lig[o0];
		o0 += blockDim.x;
	}
	if (o0 < g)
	{
		shared[o0] = lig[o0];
	}
	__syncthreads();

	// Randomize s0x.
	hiprand_init(sed, gid, 0, &crs);
	rd0 = hiprand_uniform(&crs);
	s0x[o0  = gid] = rd0 * cr1.x + (1 - rd0) * cr0.x;
	rd0 = hiprand_uniform(&crs);
	s0x[o0 += gds] = rd0 * cr1.y + (1 - rd0) * cr0.y;
	rd0 = hiprand_uniform(&crs);
	s0x[o0 += gds] = rd0 * cr1.z + (1 - rd0) * cr0.z;
	rd0 = hiprand_uniform(&crs);
	rd1 = hiprand_uniform(&crs);
	rd2 = hiprand_uniform(&crs);
	rd3 = hiprand_uniform(&crs);
	rst = rsqrtf(rd0*rd0 + rd1*rd1 + rd2*rd2 + rd3*rd3);
	s0x[o0 += gds] = rd0 * rst;
	s0x[o0 += gds] = rd1 * rst;
	s0x[o0 += gds] = rd2 * rst;
	s0x[o0 += gds] = rd3 * rst;
	for (i = 6; i < nv; ++i)
	{
		s0x[o0 += gds] = hiprand_uniform(&crs);
	}
/*
	s0x[o0  = gid] =  49.799f;
	s0x[o0 += gds] = -31.025f;
	s0x[o0 += gds] =  35.312f;
	s0x[o0 += gds] = 1.0f;
	s0x[o0 += gds] = 0.0f;
	s0x[o0 += gds] = 0.0f;
	s0x[o0 += gds] = 0.0f;
	for (i = 6; i < nv; ++i)
	{
		s0x[o0 += gds] = 0.0f;
	}
*/
	evaluate(s0e, s0g, s0a, s0q, s0c, s0d, s0f, s0t, s0x, nf, na, np, eub);

	// Mutate s0x into s1x
	o0  = gid;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
	o0 += gds;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
	o0 += gds;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
//	for (i = 3; i < nv + 1; ++i)
	for (i = 2 - nv; i < 0; ++i)
	{
		o0 += gds;
		s1x[o0] = s0x[o0];
	}
	evaluate(s1e, s1g, s1a, s1q, s1c, s1d, s1f, s1t, s1x, nf, na, np, eub);

	// Initialize the inverse Hessian matrix to identity matrix.
	// An easier option that works fine in practice is to use a scalar multiple of the identity matrix,
	// where the scaling factor is chosen to be in the range of the eigenvalues of the true Hessian.
	// See N&R for a recipe to find this initializer.
	bfh[o0 = gid] = 1.0f;
	for (j = 1; j < nv; ++j)
	{
		for (i = 0; i < j; ++i)
		{
			bfh[o0 += gds] = 0.0f;
		}
		bfh[o0 += gds] = 1.0f;
	}

	// Repeat for a number of generations.
	for (g = 0; g < nbi; ++g)
	{
		// Use BFGS to optimize the mutated conformation s1x into local optimum s2x.
		// http://en.wikipedia.org/wiki/BFGS_method
		// http://en.wikipedia.org/wiki/Quasi-Newton_method

		// Calculate p = -h * g, where p is for descent direction, h for Hessian, and g for gradient.
		sum = bfh[o1 = gid] * s1g[o0 = gid];
		for (i = 1; i < nv; ++i)
		{
			sum += bfh[o1 += i * gds] * s1g[o0 += gds];
		}
		bfp[o2 = gid] = -sum;
		for (j = 1; j < nv; ++j)
		{
			sum = bfh[o1 = (j*(j+1)>>1) * gds + gid] * s1g[o0 = gid];
			for (i = 1; i < nv; ++i)
			{
				sum += bfh[o1 += i > j ? i * gds : gds] * s1g[o0 += gds];
			}
			bfp[o2 += gds] = -sum;
		}

		// Calculate pg = p * g = -h * g^2 < 0
		o0 = gid;
		pg1 = bfp[o0] * s1g[o0];
		for (i = 1; i < nv; ++i)
		{
			o0 += gds;
			pg1 += bfp[o0] * s1g[o0];
		}
		pga = 0.0001f * pg1;
		pgc = 0.9f * pg1;

		// Perform a line search to find an appropriate alpha.
		// Try different alpha values for nls times.
		// alpha starts with 1, and shrinks to 0.1 of itself iteration by iteration.
		alp = 1.0f;
		for (j = 0; j < nls; ++j)
		{
			// Calculate x2 = x1 + a * p.
			o0  = gid;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s1xq0 = s1x[o0];
			pr0 = bfp[o0];
			o0 += gds;
			s1xq1 = s1x[o0];
			pr1 = bfp[o0];
			o0 += gds;
			s1xq2 = s1x[o0];
			pr2 = bfp[o0];
			o0 += gds;
			s1xq3 = s1x[o0];
			//assert(fabs(s1xq0*s1xq0 + s1xq1*s1xq1 + s1xq2*s1xq2 + s1xq3*s1xq3 - 1.0f) < 2e-3f);
			nrm = sqrt(pr0*pr0 + pr1*pr1 + pr2*pr2);
			ang = 0.5f * alp * nrm;
//			sng = sinf(ang) / nrm;
//			pq0 = cosf(ang);
			sincosf(ang, &sng, &pq0);
//			sincospif(ang, &sng, &pq0);
			sng /= nrm;
			pq1 = sng * pr0;
			pq2 = sng * pr1;
			pq3 = sng * pr2;
			//assert(fabs(pq0*pq0 + pq1*pq1 + pq2*pq2 + pq3*pq3 - 1.0f) < 2e-3f);
			s2xq0 = pq0 * s1xq0 - pq1 * s1xq1 - pq2 * s1xq2 - pq3 * s1xq3;
			s2xq1 = pq0 * s1xq1 + pq1 * s1xq0 + pq2 * s1xq3 - pq3 * s1xq2;
			s2xq2 = pq0 * s1xq2 - pq1 * s1xq3 + pq2 * s1xq0 + pq3 * s1xq1;
			s2xq3 = pq0 * s1xq3 + pq1 * s1xq2 - pq2 * s1xq1 + pq3 * s1xq0;
			//assert(fabs(s2xq0*s2xq0 + s2xq1*s2xq1 + s2xq2*s2xq2 + s2xq3*s2xq3 - 1.0f) < 2e-3f);
			s2x[o0 -= 3 * gds] = s2xq0;
			s2x[o0 += gds] = s2xq1;
			s2x[o0 += gds] = s2xq2;
			s2x[o0 += gds] = s2xq3;
			for (i = 6; i < nv; ++i)
			{
				bpi = bfp[o0];
				o0 += gds;
				s2x[o0] = s1x[o0] + alp * bpi;
			}

			// Evaluate x2, subject to Wolfe conditions http://en.wikipedia.org/wiki/Wolfe_conditions
			// 1) Armijo rule ensures that the step length alpha decreases f sufficiently.
			// 2) The curvature condition ensures that the slope has been reduced sufficiently.
			if (evaluate(s2e, s2g, s2a, s2q, s2c, s2d, s2f, s2t, s2x, nf, na, np, s1e[gid] + alp * pga))
			{
				o0 = gid;
				pg2 = bfp[o0] * s2g[o0];
				for (i = 1; i < nv; ++i)
				{
					o0 += gds;
					pg2 += bfp[o0] * s2g[o0];
				}
				if (pg2 >= pgc) break;
			}

			alp *= 0.1f;
		}

		// If no appropriate alpha can be found, restart the BFGS loop.
		if (j == nls)
		{
			// Accept x1 according to Metropolis criteria.
			if (s1e[gid] < s0e[gid])
			{
				o0 = gid;
				s0e[o0] = s1e[o0];
//				for (i = 1; i < nv + 2; ++i)
				for (i = -1 - nv; i < 0; ++i)
				{
					o0 += gds;
					s0e[o0] = s1e[o0];
				}
			}

			// Mutate s0x into s1x
			o0  = gid;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
			o0 += gds;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
			o0 += gds;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
//			for (i = 3; i < nv + 1; ++i)
			for (i = 2 - nv; i < 0; ++i)
			{
				o0 += gds;
				s1x[o0] = s0x[o0];
			}
			evaluate(s1e, s1g, s1a, s1q, s1c, s1d, s1f, s1t, s1x, nf, na, np, eub);

			// Initialize the inverse Hessian matrix to identity matrix.
			bfh[o0 = gid] = 1.0f;
			for (j = 1; j < nv; ++j)
			{
				for (i = 0; i < j; ++i)
				{
					bfh[o0 += gds] = 0.0f;
				}
				bfh[o0 += gds] = 1.0f;
			}
		}
		else
		{
			// Calculate y = g2 - g1.
			o0 = gid;
			bfy[o0] = s2g[o0] - s1g[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				bfy[o0] = s2g[o0] - s1g[o0];
			}

			// Calculate m = -h * y.
			sum = bfh[o1 = gid] * bfy[o0 = gid];
			for (i = 1; i < nv; ++i)
			{
				sum += bfh[o1 += i * gds] * bfy[o0 += gds];
			}
			bfm[o2 = gid] = -sum;
			for (j = 1; j < nv; ++j)
			{
				sum = bfh[o1 = (j*(j+1)>>1) * gds + gid] * bfy[o0 = gid];
				for (i = 1; i < nv; ++i)
				{
					sum += bfh[o1 += i > j ? i * gds : gds] * bfy[o0 += gds];
				}
				bfm[o2 += gds] = -sum;
			}

			// Calculate yhy = -y * m = -y * (-h * y) = y * h * y.
			o0 = gid;
			yhy = -bfy[o0] * bfm[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				yhy -= bfy[o0] * bfm[o0];
			}

			// Calculate yps = y * p.
			o0 = gid;
			yps = bfy[o0] * bfp[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				yps += bfy[o0] * bfp[o0];
			}

			// Update Hessian matrix h.
			ryp = 1.0f / yps;
			pco = ryp * (ryp * yhy + alp);
			o2 = gid;
			for (j = 0; j < nv; ++j)
			{
				bpj = bfp[o2];
				bmj = bfm[o2];
				ppj = pco * bpj;
				bfh[o1 = (j*(j+3)>>1) * gds + gid] += (ryp * 2 * bmj + ppj) * bpj;
				for (i = j + 1; i < nv; ++i)
				{
					o0 = i * gds + gid;
					bpi = bfp[o0];
					bfh[o1 += i * gds] += ryp * (bmj * bpi + bfm[o0] * bpj) + ppj * bpi;
				}
				o2 += gds;
			}

			// Move to the next iteration, i.e. e1 = e2, x1 = x2, g1 = g2.
			o0 = gid;
			s1e[o0] = s2e[o0];
//			for (i = 1; i < 2 * (nv + 1); ++i)
			for (i = -1 - 2 * nv; i < 0; ++i)
			{
				o0 += gds;
				s1e[o0] = s2e[o0];
			}
		}
	}

	// Accept x1 according to Metropolis criteria.
	if (s1e[gid] < s0e[gid])
	{
		o0 = gid;
		s0e[o0] = s1e[o0];
//		for (i = 1; i < nv + 2; ++i)
		for (i = -1 - nv; i < 0; ++i)
		{
			o0 += gds;
			s0e[o0] = s1e[o0];
		}
	}
}
